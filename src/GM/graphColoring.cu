#include "hip/hip_runtime.h"
#include "graphColoring.h"



//----------------------- SDO improved -----------------------//
//
// Author: Shusen & Pascal
// returns the degree of that node
int __device__ degree(int vertex, int *degreeList){
	return degreeList[vertex];
}



// Author: Shusen & Pascal
// saturation of a vertex
int __device__ saturation(int vertex, int *adjacencyList, int *graphColors, int maxDegree, int start, int end){
	int saturation = 0;	
	int colors[TEMP_COLOR_LENGTH];			
	for (int j=0; j<TEMP_COLOR_LENGTH; j++)	// OPTION2  
	//for (int j=0; j<(maxDegree+1); j++)  		// OPTION1
		colors[j] = 0;
	
	
	for (int i=0; i<maxDegree; i++){
		if (adjacencyList[vertex*maxDegree + i] < start)
			continue;
		
		if (adjacencyList[vertex*maxDegree + i] > end)
			break;
		
		if (adjacencyList[vertex*maxDegree + i] != -1)
			//colors[ graphColors[vertex] ] = 1;			// at each colored set the array to 1
			colors[ graphColors[adjacencyList[vertex*maxDegree + i]] ] = 1;			// at each colored set the array to 1
		else
			break;
	}
	
	// count the number of 1's but skip uncolored
	for (int i=1; i<TEMP_COLOR_LENGTH; i++)		// OPTION2
	//for (int i=1; i<maxDegree+1; i++)			// OPTION1
		if (colors[i] == 1)
			saturation++;
	
	return saturation;
}




// Author: Shusen & Pascal
// colors the vertex with the min possible color
int __device__ color(int vertex, int *adjacencyList, int *graphColors, int maxDegree, int numColored, int start, int end, int disp){
	int colors[TEMP_COLOR_LENGTH];			
	for (int j=0; j<TEMP_COLOR_LENGTH; j++)	// OPTION2
	//for (int j=0; j<(maxDegree+1); j++)		// OPTION1
		colors[j] = 0;
	
	
	if (graphColors[vertex] == 0)
		numColored++;
	
	for (int i=0; i<maxDegree; i++){						// set the index of the color to 1	
		// Limits color checking to subgraph
		/*
		 if (adjacencyList[vertex*maxDegree + i] < start)
		 continue;
		 
		 if (adjacencyList[vertex*maxDegree + i] > end)
		 break;
		 */
		
		if (adjacencyList[vertex*maxDegree + i] != -1)
			colors[  graphColors[  adjacencyList[vertex*maxDegree + i]  ]  ] = 1;
		else 
			break;
	}
	
	
	// nodes still equal to 0 are unassigned
	for (int i=1; i<TEMP_COLOR_LENGTH; i++)		// OPTION2	
	//for (int i=1; i<maxDegree+1; i++)				// OPTION1				
		if (colors[i] != 1){
			if (disp == 0){
				graphColors[vertex] = i;
				break;
			}
			else
				disp--;
		}
	
	return numColored;
}





// Author: Shusen & Pascal
// does the coloring
__global__ void colorGraph_SDO(int *adjacencyList, int *graphColors, int *degreeList, int sizeGraph, int maxDegree, 
								int *startPartitionListD, int *endPartitionListD, int *randomListD)
{
	int start, end, partitionIndex;
	int subGraphSize, numColored = 0;
	int satDegree, max, index;
	int randomCount = 0;
	
	//subGraphSize = sizeGraph/(gridDim.x * blockDim.x);
	//start = (sizeGraph/gridDim.x * blockIdx.x) + (subGraphSize * threadIdx.x);
	//end = start + subGraphSize;
	
	partitionIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
	start = startPartitionListD[partitionIndex];
	end = endPartitionListD[partitionIndex];
	subGraphSize = end - start;
	
	while (numColored < subGraphSize){
		randomCount++;
		randomCount = randomCount%10;
		
		max = -1;
		
		for (int i=start; i<end; i++){
			if (graphColors[i] == 0)			// not colored
			{
				satDegree = saturation(i,adjacencyList,graphColors, maxDegree, start, end);
				
				if (satDegree > max){
					max = satDegree;
					index = i;				
				}
				
				if (satDegree == max){
					if (degree(i,degreeList) > degree(index,degreeList))
						index = i;
				}
			}
			
	//			if (graphColors[index] == 0)
	//	    		numColored = color(index,adjacencyList,graphColors, maxDegree, numColored, start, end, randomListD[partitionIndex*10 + randomCount]);
		}
		numColored = color(index,adjacencyList,graphColors, maxDegree, numColored, start, end, randomListD[partitionIndex*10 + randomCount]);
	}
}

__global__ void colorGraph_OMax(int *adjacencyList, int *graphColors, int *degreeList, int sizeGraph, int maxDegree,
                             int *startPartitionListD, int *endPartitionListD, int *randomListD, int *numOutD)
{
        int start, end, partitionIndex;
        int subGraphSize, numColored = 0;
        int max, index, numOut;
        int randomCount = 0;

        //subGraphSize = sizeGraph/(gridDim.x * blockDim.x);
        //start = (sizeGraph/gridDim.x * blockIdx.x) + (subGraphSize * threadIdx.x);
        //end = start + subGraphSize;

        partitionIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
        start = startPartitionListD[partitionIndex];
        end = endPartitionListD[partitionIndex];
        subGraphSize = end - start;

        while (numColored < subGraphSize){
                randomCount++;
                randomCount = randomCount%10;

                max = -1;

                for (int i=start; i<end; i++){
                        if (graphColors[i] == 0)                        // not colored
                        {
								numOut = numOutD[i];

								if (numOut > max){
									max = numOut;
									index = i;
								}

				
								if (numOut == max){
                                        if (degree(i,degreeList) > degree(index,degreeList))
                                                index = i;
                                }
                        }
						
						if (graphColors[index] == 0)
                      		numColored = color(index,adjacencyList,graphColors, maxDegree, numColored, start, end, randomListD[partitionIndex*10 + randomCount]);
                }
                //numColored = color(index,adjacencyList,graphColors, maxDegree, numColored, start, end, randomListD[partitionIndex*10 + randomCount]);
        }
}


__global__ void colorGraph_OMin(int *adjacencyList, int *graphColors, int *degreeList, int sizeGraph, int maxDegree,
                             int *startPartitionListD, int *endPartitionListD, int *randomListD, int *numOutD)
{
        int start, end, partitionIndex;
        int subGraphSize, numColored = 0;
        int index, numOut, min;
        int randomCount = 0;

        //subGraphSize = sizeGraph/(gridDim.x * blockDim.x);
        //start = (sizeGraph/gridDim.x * blockIdx.x) + (subGraphSize * threadIdx.x);
        //end = start + subGraphSize;

        partitionIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
        start = startPartitionListD[partitionIndex];
        end = endPartitionListD[partitionIndex];
        subGraphSize = end - start;

        while (numColored < subGraphSize){
                randomCount++;
                randomCount = randomCount%10;

				min = 100000;

                for (int i=start; i<end; i++){
                        if (graphColors[i] == 0)                        // not colored
                        {
								numOut = numOutD[i];

								if (numOut < min){
									min = numOut;
									index = i;
								}

				
								if (numOut == min){
                                        if (degree(i,degreeList) > degree(index,degreeList))
                                                index = i;
                                }
                        }

						if (graphColors[index] == 0)
                      		numColored = color(index,adjacencyList,graphColors, maxDegree, numColored, start, end, randomListD[partitionIndex*10 + randomCount]);
                }
                //numColored = color(index,adjacencyList,graphColors, maxDegree, numColored, start, end, randomListD[partitionIndex*10 + randomCount]);
        }
}


__global__ void conflictSolveMIN(int *adjacencyList, int *conflict, int *graphColors, int *degreeList, 
			       	int sizeGraph, int maxDegree, int *startPartitionListD, int *endPartitionListD, int *randomListD, int *numOutD){
	int start, end, index, partitionIndex;
	int numColored = 0;
	int min, numOut;
	int randomCount = 0;
	int numOfInitialConflicts = 0;
	
	
	// int subGraphSize;
	//subGraphSize = sizeGraph/(gridDim.x * blockDim.x);
	//start = (sizeGraph/gridDim.x * blockIdx.x) + (subGraphSize * threadIdx.x);
	//end = start + subGraphSize;
	
	partitionIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
	start = startPartitionListD[partitionIndex];
	end = endPartitionListD[partitionIndex];
	
	
	
	// Count the number of conflicts
	for (int i=start; i<end; i++)
		if (graphColors[i] == 0)
			numOfInitialConflicts++;
    
	
    	while (numOfInitialConflicts > numColored){
        	min = 100000;
        	randomCount++;
			randomCount = randomCount%10;
        
        	for (int i=start; i<end; i++){
            	if (graphColors[i] == 0)                        // not colored
            	{
				numOut = numOutD[i];

				if (numOut < min){
					min = numOut;
					index = i;
				}

				
				if (numOut == min){
                		if (degree(i,degreeList) > degree(index,degreeList))
                     	index = i;
                }
				
				if (graphColors[index] == 0)
					numColored = color(index,adjacencyList,graphColors, maxDegree, numColored, start, end, randomListD[partitionIndex*10 + randomCount]);
            }
			//numColored = color(index,adjacencyList,graphColors, maxDegree, numColored, start, end, randomListD[partitionIndex*10 + randomCount]);
        }
    }
}




__global__ void conflictSolveMAX(int *adjacencyList, int *conflict, int *graphColors, int *degreeList, 
			       	int sizeGraph, int maxDegree, int *startPartitionListD, int *endPartitionListD, int *randomListD, int *numOutD){
	int start, end, index, partitionIndex;
	int numColored = 0;
	int max, numOut;
	int randomCount = 0;
	int numOfInitialConflicts = 0;
	
	
	// int subGraphSize;
	//subGraphSize = sizeGraph/(gridDim.x * blockDim.x);
	//start = (sizeGraph/gridDim.x * blockIdx.x) + (subGraphSize * threadIdx.x);
	//end = start + subGraphSize;
	
	partitionIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
	start = startPartitionListD[partitionIndex];
	end = endPartitionListD[partitionIndex];
	
	
	
	// Count the number of conflicts
	for (int i=start; i<end; i++)
		if (graphColors[i] == 0)
			numOfInitialConflicts++;
    
	
    	while (numOfInitialConflicts > numColored){
        	max = -1;
        	randomCount++;
		randomCount = randomCount%10;
        
        	for (int i=start; i<end; i++){
            	if (graphColors[i] == 0)                        // not colored
            	{
				numOut = numOutD[i];

				if (numOut > max){
					max = numOut;
					index = i;
				}

				
				if (numOut == max){
                		if (degree(i,degreeList) > degree(index,degreeList))
                     	index = i;
                }
				
				if (graphColors[index] == 0)
					numColored = color(index,adjacencyList,graphColors, maxDegree, numColored, start, end, randomListD[partitionIndex*10 + randomCount]);
            	}
			//numColored = color(index,adjacencyList,graphColors, maxDegree, numColored, start, end, randomListD[partitionIndex*10 + randomCount]);
        }
    }
}


//Author: Pascal
//recolors nodes where we have conflicts
__global__ void conflictSolveSDO(int *adjacencyList, int *conflict, int *graphColors, int *degreeList, 
			       	int sizeGraph, int maxDegree, int *startPartitionListD, int *endPartitionListD, int *randomListD){
	int start, end, index, partitionIndex;
	int numColored = 0;
	int satDegree, max;
	int randomCount = 0;
	int numOfInitialConflicts = 0;
	
	
	// int subGraphSize;
	//subGraphSize = sizeGraph/(gridDim.x * blockDim.x);
	//start = (sizeGraph/gridDim.x * blockIdx.x) + (subGraphSize * threadIdx.x);
	//end = start + subGraphSize;
	
	partitionIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
	start = startPartitionListD[partitionIndex];
	end = endPartitionListD[partitionIndex];
	
	
	
	// Count the number of conflicts
	for (int i=start; i<end; i++)
		if (graphColors[i] == 0)
			numOfInitialConflicts++;
    
	
    while (numOfInitialConflicts > numColored){
        max = -1;
        randomCount++;
		randomCount = randomCount%10;
        
        for (int i=start; i<end; i++){
            if (graphColors[i] == 0)                        // not colored
            {
				satDegree = saturation(i,adjacencyList,graphColors, maxDegree, start, end);
				
                if (satDegree > max){
                    max = satDegree;
                    index = i;                              
                }
				
                if (satDegree == max){
                    if (degree(i,degreeList) > degree(index,degreeList))
                        index = i;
                }
				
			//	if (graphColors[index] == 0)
			//		numColored = color(index,adjacencyList,graphColors, maxDegree, numColored, start, end, randomListD[partitionIndex*10 + randomCount]);
            }

			numColored = color(index,adjacencyList,graphColors, maxDegree, numColored, start, end, randomListD[partitionIndex*10 + randomCount]);
        }
    }
}





//----------------------- First Fit Adjacency List -----------------------//
//
// Author: Pascal
// First Fit
__global__ void colorGraph_FF(int *adjacencyListD, int *colors, int size, int maxDegree, int *startPartitionListD, int *endPartitionListD){
	int i, start, end, partitionIndex;
	
	int tempColors[TEMP_COLOR_LENGTH];
	
	//int subGraphSize;
	//subGraphSize = size/(gridDim.x * blockDim.x);
	//start = (size/gridDim.x * blockIdx.x) + (subGraphSize * threadIdx.x);
	//end = start + subGraphSize;
	
	partitionIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
	start = startPartitionListD[partitionIndex];
	end = endPartitionListD[partitionIndex];
	
	
	for (i=start; i<end; i++)
	{
		for (int j=0; j<TEMP_COLOR_LENGTH; j++)		// OPTION2	
		//for (int j=0; j<maxDegree; j++)					// OPTION1
			tempColors[j] = 0;
		
		
		for (int j=0; j<maxDegree; j++){
			int vertexNeigh = i*maxDegree + j;
			
			if (adjacencyListD[vertexNeigh] == -1)
				break;
			else
				tempColors[ colors[adjacencyListD[vertexNeigh]] ] = 1;
		}
		
		
		for (int j=1; j<TEMP_COLOR_LENGTH; j++)		// OPTION2	
		//for (int j=1; j<maxDegree; j++)					// OPTION1	
			if (tempColors[j] != 1){
				colors[i] = j;
				break;
			}	
	}
}





__global__ void recolorGraph_FF(int *adjacencyListD, int *colors, int size, int maxDegree, int *startPartitionListD, int *endPartitionListD){
	int i, start, end, partitionIndex;
	
	int tempColors[TEMP_COLOR_LENGTH];
	
	//int subGraphSize;
	//subGraphSize = size/(gridDim.x * blockDim.x);
	//start = (size/gridDim.x * blockIdx.x) + (subGraphSize * threadIdx.x);
	//end = start + subGraphSize;
	
	partitionIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
	start = startPartitionListD[partitionIndex];
	end = endPartitionListD[partitionIndex];
	
	
	
	
	for (i=start; i<end; i++)
	{
		if (colors[i] != 0)		// skip if already colored
			continue;
			
		for (int j=0; j<TEMP_COLOR_LENGTH; j++)		// OPTION2	
		//for (int j=0; j<maxDegree; j++)					// OPTION1	
			tempColors[j] = 0;
		
		
		for (int j=0; j<maxDegree; j++){
			int vertexNeigh = i*maxDegree + j;
			
			if (adjacencyListD[vertexNeigh] == -1)
				break;
			else
				tempColors[ colors[adjacencyListD[vertexNeigh]] ] = 1;
		}
		
		
		for (int j=1; j<TEMP_COLOR_LENGTH; j++)		// OPTION2	
		//for (int j=1; j<maxDegree; j++)					// OPTION1	
			if (tempColors[j] != 1){
				colors[i] = j;
				break;
			}	
	}
}



//----------------------- Detects conflicts -----------------------//
//
// Author: Peihong
// each thread deals with 1 vertex from boundary list
// 		set the conflicted color to 0
// 		set its value in the conflict list to point to the node

/* 冲突检测（解决） */
__global__ void conflictsDetection(int *adjacentListD, int *boundaryListD, int *colors, int *conflictD, long size, int boundarySize, int maxDegree){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int nodeFrom, nodeTo;
	
	
	if (idx < boundarySize){
		nodeFrom = boundaryListD[idx];
		

		for (int i=0; i<maxDegree; i++)
		{
			nodeTo = adjacentListD[nodeFrom*maxDegree + i];
			
			if (nodeTo == -1)
				break;
			
			if (nodeFrom>=nodeTo && (colors[nodeFrom] == colors[nodeTo]))	/* 如果存在着相同颜色的邻接点且顶点号比当前结点大，则将当前结点作为冲突结点 */
			{
				conflictD[idx] = nodeFrom;	
				/* 冲突结点颜色置0 */
				colors[nodeFrom] = 0;				// added!!!!!!!!
			}		
		}
	}
}




//----------------------- Main -----------------------//
//extern "C"
float cudaGraphColoring(int *adjacentList, int *boundaryList, int *graphColors, int *degreeList, int *conflict, int boundarySize, int maxDegree, int graphSize, int & passes, int subsizeBoundary, int _gridSize, int _blockSize, int *startPartitionList, int *endPartitionList, int *randomList, int numRand, int useSDO, int *numOut)
{
	int *numOutD, *adjacentListD, *colorsD, *boundaryListD, *degreeListD, *conflictListD, *startPartitionListD, *endPartitionListD, *randomListD; 

	/* 冲突检测kernel的线程块数和线程块大小 */
	int gridsize = ceil((float)boundarySize/(float)(CONFLICT_BLOCK_SIZE));
	int blocksize = CONFLICT_BLOCK_SIZE;	/* =256 */
	int *numConflicts;
	
	hipEvent_t start_col, start_confl, stop_col, stop_confl, start_mem, stop_mem;         
    float elapsedTime_memory, elapsedTime_col, elapsedTime_confl; 
    int conflictCount = 0;	
	//int *tempColor = (int*)malloc(boundarySize * sizeof(int));	
	
	int conflictsStop = 200;
	
	//-------------- memory transfer -----------------!
	hipEventCreate(&start_mem); 
    hipEventCreate(&stop_mem); 
    hipEventRecord(start_mem, 0); 
	
	
	hipMalloc((void**)&adjacentListD, graphSize*maxDegree*sizeof(int));
	hipMalloc((void**)&colorsD, graphSize*sizeof(int));
	hipMalloc((void**)&boundaryListD, boundarySize*sizeof(int));
	hipMalloc((void**)&degreeListD, graphSize*sizeof(int));
	
	hipMalloc((void**)&numConflicts, 1*sizeof(int));
	hipMalloc((void**)&conflictListD, boundarySize*sizeof(int));
	
	hipMalloc((void**)&startPartitionListD, _gridSize*_blockSize*sizeof(int));
	hipMalloc((void**)&endPartitionListD, _gridSize*_blockSize*sizeof(int));
	hipMalloc((void**)&randomListD, numRand*sizeof(int));
	hipMalloc((void**)&numOutD, graphSize*sizeof(int));	
	
	
	hipMemcpy(adjacentListD, adjacentList, graphSize*maxDegree*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(colorsD, graphColors, graphSize*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(boundaryListD, boundaryList, boundarySize*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(degreeListD, degreeList, graphSize*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(startPartitionListD, startPartitionList, _gridSize*_blockSize*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(endPartitionListD, endPartitionList, _gridSize*_blockSize*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(randomListD, randomList, numRand*sizeof(int), hipMemcpyHostToDevice);
 	hipMemcpy(numOutD, numOut, graphSize*sizeof(int), hipMemcpyHostToDevice);
	

	hipEventRecord(stop_mem, 0);
    hipEventSynchronize(stop_mem);


	hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
    	// print the CUDA error message and exit
        cout << "Cuda error - After memory allocation: " << error << endl;
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }	
	
	/* 设置colorGraph_FF的线程块 */
	dim3 dimGrid_col(_gridSize);
	dim3 dimBlock_col(_blockSize);
	
	/* 设置conflictDetection的线程块 */
	dim3 dimGrid_confl(gridsize);
	dim3 dimBlock_confl(blocksize);
	
	
	
	
	//-------------- Sequential Graph coloring -----------------!
	hipEventCreate(&start_col); 
    hipEventCreate(&stop_col); 
    hipEventRecord(start_col, 0); 
	
	/* 调用着色kernel */
	if (useSDO == 0){	/* useSDO为0，用FF */
		colorGraph_FF<<<dimGrid_col, dimBlock_col>>>(adjacentListD, colorsD, graphSize, maxDegree, startPartitionListD, endPartitionListD);
	}
	else
		if (useSDO == 1){
			colorGraph_SDO<<<dimGrid_col, dimBlock_col>>>(adjacentListD, colorsD, degreeListD, graphSize, maxDegree, startPartitionListD, endPartitionListD, randomListD);
		}
		else
			if (useSDO == 2){
				colorGraph_OMax<<<dimGrid_col, dimBlock_col>>>(adjacentListD, colorsD, degreeListD, graphSize, maxDegree, startPartitionListD, endPartitionListD, randomListD, numOutD);
			}
			else{
				colorGraph_OMin<<<dimGrid_col, dimBlock_col>>>(adjacentListD, colorsD, degreeListD, graphSize, maxDegree, startPartitionListD, endPartitionListD, randomListD, numOutD);	
			}
	
	hipEventRecord(stop_col, 0); 
    hipEventSynchronize(stop_col);	
	/* */
	float elapsedTime_col1;
	hipEventElapsedTime(&elapsedTime_col1, start_col, stop_col);
	//printf("elapsedTime_col1=%f\n", elapsedTime_col1);
	
	

	hipError_t error1 = hipGetLastError();
  	if(error1 != hipSuccess)
  	{
		cout << "Cuda error - after kernel call: " << error1 << endl;
    		printf("CUDA error: %s\n", hipGetErrorString(error1));
    		exit(-1);
  	}

	
	hipEventCreate(&start_confl); 
    hipEventCreate(&stop_confl); 
    hipEventRecord(start_confl, 0); 
	
	/* GPU冲突检测 */
	hipMemset(conflictListD, -1, boundarySize*sizeof(int));
	conflictsDetection<<<dimGrid_confl, dimBlock_confl>>>(adjacentListD, boundaryListD, colorsD, conflictListD, graphSize, boundarySize, maxDegree);
	
	hipEventRecord(stop_confl, 0); 
    hipEventSynchronize(stop_confl); 
	/*  */
	float elapsedTime_confl1;
	hipEventElapsedTime(&elapsedTime_confl1, start_confl, stop_confl); 


	/*
	hipMemcpy(tempColor, colorsD, graphSize*sizeof(int), hipMemcpyDeviceToHost);
	conflictCount = 0;
	for (int k=0; k<graphSize; k++)
		if (tempColor[k] == 0)
			conflictCount++;

	cout << endl << "Conflicts: " << conflictCount << endl;
	if (passes == 0){
		if (conflictCount < 3000)
			passes = 2;
		else
			if (conflictCount < 10000)
				passes = 3;
			else
				passes = 4 + ((int)((conflictCount - 10000)/10000));
	}
	cout << "Passes: " << passes << endl;	
	*/


	int setPassNum = 1;
	if (passes == 0)	/* passes为0，将setPassNum置0 */
		setPassNum = 0;
	

	hipEvent_t start_memcon, stop_memcon;
    float elapsedTime_memcon;

	hipEventCreate(&start_memcon);
    hipEventCreate(&stop_memcon);
    hipEventRecord(start_memcon, 0);	

/*
	if (setPassNum == 0){
		hipMemcpy(tempColor, colorsD, graphSize*sizeof(int), hipMemcpyDeviceToHost);
    	conflictCount = 0;
    	for (int k=0; k<graphSize; k++)
        	if (tempColor[k] == 0){
            	conflictCount++;
				if (conflictCount > 200){
					passes = 2;
					break;
				}
			}
	}
*/
	/* 统计冲突结点数 */
	if (setPassNum == 0){
        //hipMemcpy(tempColor, conflictListD, boundarySize*sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(conflict, conflictListD, boundarySize*sizeof(int), hipMemcpyDeviceToHost);
        conflictCount = 0;
        for (int k=0; k<boundarySize; k++)
            if (conflict[k] != -1){
                conflictCount++;
                if (conflictCount > 200){	/* 如果冲突结点大于200，将passes置为2 */
                    passes = 2;
                    break;
                }
            }
    }

	hipEventRecord(stop_memcon, 0);
    hipEventSynchronize(stop_memcon);
	float elapsedTime_memcon1;
	hipEventElapsedTime(&elapsedTime_memcon1, start_memcon, stop_memcon);
	//cout << "Conflict count time: " << elapsedTime_memcon << endl;


	float elapsedTime_col2;
	/* 重复，直到冲突结点数小于200 */
	for (int times=1; times<passes; times++){
        hipEventCreate(&start_col);
       	hipEventCreate(&stop_col);
        hipEventRecord(start_col, 0);
         
		/* 冲突解决 */
		if (useSDO == 1)
            conflictSolveSDO<<<dimGrid_col, dimBlock_col>>>(adjacentListD, conflictListD, colorsD, degreeListD, graphSize, maxDegree, startPartitionListD, endPartitionListD, randomListD);
        else
			if (useSDO ==2)
				conflictSolveMAX<<<dimGrid_col, dimBlock_col>>>(adjacentListD, conflictListD, colorsD, degreeListD, graphSize, maxDegree, startPartitionListD, endPartitionListD, randomListD,numOutD);
			else
				if (useSDO == 3)
					conflictSolveMIN<<<dimGrid_col, dimBlock_col>>>(adjacentListD, conflictListD, colorsD, degreeListD, graphSize, maxDegree, startPartitionListD, endPartitionListD, randomListD,numOutD);
                else	/* useSDO为0, 再次调用FF */
					recolorGraph_FF<<<dimGrid_col, dimBlock_col>>>(adjacentListD, colorsD, graphSize, maxDegree, startPartitionListD, endPartitionListD);
                
		hipEventRecord(stop_col, 0);
        hipEventSynchronize(stop_col);
		/* */
		float elapsedTime_col2;
		hipEventElapsedTime(&elapsedTime_col2, start_col, stop_col);
		//printf("elapsedTime_col2=%f\n", elapsedTime_col2);
		elapsedTime_col1 += elapsedTime_col2;
		


		hipEventCreate(&start_confl);
        hipEventCreate(&stop_confl);
        hipEventRecord(start_confl, 0);

		/* 最后再调用conflictsDetection */
        hipMemset(conflictListD, -1, boundarySize*sizeof(int));
        conflictsDetection<<<dimGrid_confl, dimBlock_confl>>>(adjacentListD, boundaryListD, colorsD, conflictListD, graphSize, boundarySize, maxDegree);

        hipEventRecord(stop_confl, 0);
        hipEventSynchronize(stop_confl);
		/*  */
		float elapsedTime_confl2;
		hipEventElapsedTime(&elapsedTime_confl2, start_confl, stop_confl);
		elapsedTime_confl1 += elapsedTime_confl2;
	

		hipEventCreate(&start_memcon);
    	hipEventCreate(&stop_memcon);
    	hipEventRecord(start_memcon, 0);
		/*
		if (setPassNum == 0){
			hipMemcpy(tempColor, colorsD, graphSize*sizeof(int), hipMemcpyDeviceToHost);
			conflictCount = 0;
        	for (int k=0; k<graphSize; k++)
            	if (tempColor[k] == 0){
                	conflictCount++;
                	if (conflictCount > 200){
                    	passes++;
                    	break;
                	}
            	}
		}
		*/
	
		/* 统计冲突结点数 */
		if (setPassNum == 0){        
		//	hipMemcpy(tempColor, conflictListD, boundarySize*sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(conflict, conflictListD, boundarySize*sizeof(int), hipMemcpyDeviceToHost);
			conflictCount = 0;
        	for (int k=0; k<boundarySize; k++)
            	if (conflict[k] != -1){
                	conflictCount++;
                	if (conflictCount > conflictsStop){	/* 大于200，则将passes加1 */
                    	passes++;
                    	break;
                	}
            	}
    	}

		hipEventRecord(stop_memcon, 0);
    	hipEventSynchronize(stop_memcon);
		float elapsedTime_memcon2;
    	hipEventElapsedTime(&elapsedTime_memcon2, start_memcon, stop_memcon);
		elapsedTime_memcon1 += elapsedTime_memcon2;
    	//cout << "Conflict count time: " << elapsedTime_memcon << endl;


	}
	
	//cout << "Passes done: " << passes << endl;

	//-------------- Cleanup -----------------!
	hipMemcpy(graphColors, colorsD, graphSize*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(conflict, conflictListD, boundarySize*sizeof(int), hipMemcpyDeviceToHost);
	
	
	
	hipEventElapsedTime(&elapsedTime_memory, start_mem, stop_mem); 
	
	hipEventElapsedTime(&elapsedTime_col, start_col, stop_col);
	cout << "elapsedTime_col: " << elapsedTime_col << " ms" << "elapsedTime_col1: " << elapsedTime_col1 << " ms" << endl;
	elapsedTime_col = elapsedTime_col1;
	
	hipEventElapsedTime(&elapsedTime_confl, start_confl, stop_confl);
	cout << "elapsedTime_confl: " << elapsedTime_confl << " ms" << "elapsedTime_confl1: " << elapsedTime_confl1 << " ms" << endl;
	elapsedTime_confl = elapsedTime_confl1;
	
	elapsedTime_memcon = elapsedTime_memcon1;
	cout << "Conflict count time: " << elapsedTime_memcon << " ms" << endl;
	
	cout << endl << "GPU timings ~ Memory transfer: " << elapsedTime_memory  << " ms     Coloring: " << elapsedTime_col << " ms    Conflict: " << elapsedTime_confl << " ms" << endl; 
	
//	delete []tempColor;
	
	hipFree(adjacentListD);
	hipFree(colorsD);
	hipFree(boundaryListD);
	hipFree(degreeListD);
	hipFree(numConflicts);
	hipFree(conflictListD);
	hipFree(startPartitionListD);
	hipFree(endPartitionListD);
	hipFree(randomListD);
	
	return (elapsedTime_col + elapsedTime_confl + elapsedTime_memcon);
}

